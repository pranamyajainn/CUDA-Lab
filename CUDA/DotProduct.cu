#include "hip/hip_runtime.h"
%%cuda
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 256

__global__ void vectorMultiply(int *a, int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] * b[i];
    }
}

int main() {
    int n = N;

    int *a = (int *)malloc(n * sizeof(int));
    int *b = (int *)malloc(n * sizeof(int));
    int *c = (int *)malloc(n * sizeof(int));

    for (int i = 0; i < n; i++) {
        a[i] = i + 1;
        b[i] = (i + 1) * 2;
    }

    int *d_a, *d_b, *d_c;

    hipMalloc((void **)&d_a, n * sizeof(int));
    hipMalloc((void **)&d_b, n * sizeof(int));
    hipMalloc((void **)&d_c, n * sizeof(int));

    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, n * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    vectorMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    hipDeviceSynchronize();

    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    int dotProduct = 0;
    for (int i = 0; i < n; i++) {
        dotProduct += c[i];
    }

    printf("Dot Product: %d\n", dotProduct);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}
