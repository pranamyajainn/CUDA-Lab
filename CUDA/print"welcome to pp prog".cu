
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel to print a message
__global__ void printMessage() {
    int threadId = threadIdx.x + blockIdx.x * blockDim.x; // Unique thread ID
    printf("Thread %d: Welcome to Parallel Programming!\n", threadId);
}

int main() {
    int threads, blocks; // User-defined threads and blocks

    // Get user input
    printf("Enter the number of threads per block: ");
    scanf("%d", &threads);
    printf("Enter the number of blocks per grid: ");
    scanf("%d", &blocks);

    // Launch the kernel with user-defined configuration
    printMessage<<<blocks, threads>>>();

    // Wait for GPU to finish before exiting
    hipDeviceSynchronize();

    return 0;
}
