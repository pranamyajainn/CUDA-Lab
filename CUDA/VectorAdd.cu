#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Unique thread ID
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int n;
    printf("Enter the size of the vectors: ");
    scanf("%d", &n);

    int *a = (int *)malloc(n * sizeof(int));
    int *b = (int *)malloc(n * sizeof(int));
    int *c = (int *)malloc(n * sizeof(int));

    for (int i = 0; i < n; i++) {
        a[i] = i + 1;
        b[i] = (i + 1) * 2;
    }

    int *d_a, *d_b, *d_c;

    // Allocate memory on the device
    hipMalloc((void **)&d_a, n * sizeof(int));
    hipMalloc((void **)&d_b, n * sizeof(int));
    hipMalloc((void **)&d_c, n * sizeof(int));

    // Copy input vectors to the device
    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Vector Addition Result:\n");
    for (int i = 0; i < n; i++) {
        printf("%d ", c[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(a);
    free(b);
    free(c);

    return 0;
}
