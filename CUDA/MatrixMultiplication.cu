#include "hip/hip_runtime.h"
%%cuda
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 32

__global__ void matrixMultiply(int *d_A, int *d_B, int *d_C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        int sum = 0;
        for (int k = 0; k < n; k++) {
            sum += d_A[row * n + k] * d_B[k * n + col];
        }
        d_C[row * n + col] = sum;
    }
}

int main() {
    int A[N][N], B[N][N], C[N][N];
    int *d_A, *d_B, *d_C;

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i][j] = i + j;
            B[i][j] = i - j;
        }
    }

    size_t size = N * N * sizeof(int);
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(3, 3);
    dim3 blocksPerGrid(1, 1);

    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    printf("Result matrix:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
