
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel to add two numbers
__global__ void addNumbers(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main() {
    int a = 5, b = 3, c;
    int *d_a, *d_b, *d_c;

    // Allocate memory on the device
    hipMalloc((void **)&d_a, sizeof(int));
    hipMalloc((void **)&d_b, sizeof(int));
    hipMalloc((void **)&d_c, sizeof(int));

    // Copy input values to the device
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel
    addNumbers<<<1, 1>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("Result: %d + %d = %d\n", a, b, c);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
