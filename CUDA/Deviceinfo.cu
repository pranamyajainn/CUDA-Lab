
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device %d: %s\n", i, prop.name);
        printf("  Total Global Memory: %lu bytes\n", prop.totalGlobalMem);
        printf("  Multiprocessors: %d\n", prop.multiProcessorCount);
        printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
    }

    return 0;
}
